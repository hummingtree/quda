#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <gauge_field_order.h>

#include <mdw_dslash5_tensor_core.cuh>

namespace quda {
  namespace mobius_tensor_core {

#if defined(GPU_DOMAIN_WALL_DIRAC) && (__COMPUTE_CAPABILITY__ >= 700) && (__COMPUTE_CAPABILITY__ <= 750)
    
    
    /**
      @brief Parameter structure for applying the Dslash
    */
    template <class storage_type, int Ls_> // storage_type is the usual "Float" in other places in QUDA
    struct FusedDslashArg {
      static constexpr bool spin_project = true;
      static constexpr bool spinor_direct_load = false; // false means texture load
      typedef typename colorspinor_mapper<storage_type, 4, 3, spin_project, spinor_direct_load>::type
          F; // color spin field order
      typedef typename mapper<storage_type>::type real; // the compute type for the in kernel computation
      static constexpr bool gauge_direct_load = false; // false means texture load
      static constexpr QudaGhostExchange ghost = QUDA_GHOST_EXCHANGE_PAD;
      typedef typename gauge_mapper<storage_type, QUDA_RECONSTRUCT_NO, 18, QUDA_STAGGERED_PHASE_NO, gauge_direct_load,
          ghost>::type G; // gauge field order

      F out; // output vector field
      const F in; // input vector field
      F y; // auxiliary output vector field
      const F x; // auxiliary input vector field

      const G U; // The gauge field

      const int nParity; // number of parities we're working on
      const int parity; // output parity of this dslash operator
      const int volume_cb; // checkerboarded volume
      const int volume_4d_cb; // 4-d checkerboarded volume

      const int dim[4];

      const int_fastdiv Ls; // length of 5th dimension

      const int shift[4]; // sites where we actually calculate.
      const int halo_shift[4]; // halo means zero. When we are expanding we have halo of cs-field where values are zero.

      const int_fastdiv shrinked_dim[4]; // dimension after shifts are considered.

      // partial kernel and expansion parameters
      const int volume_4d_cb_shift; // number of 4d sites we need calculate
      // const int volume_4d_cb_expansive; //

      const real m_f; // fermion mass parameter
      const real m_5; // Wilson mass shift

      const bool dagger; // dagger
      //    const bool xpay;        // whether we are doing xpay or not

      real b; // real constant Mobius coefficient
      real c; // real constant Mobius coefficient
      real a; // real xpay coefficient

      real kappa;
      real fac_inv;

      // (beta + alpha*m5inv) @ in
      real alpha = 1.;
      real beta = 0.;

      const float scale; // scale factor for the input color spin field
      real m_scale = 1.; // scale factor for the matrix

      MdwfFusedDslashType type;
      FusedDslashArg(ColorSpinorField& out, const ColorSpinorField& in, const GaugeField& U, ColorSpinorField& y,
          const ColorSpinorField& x, double m_f_, double m_5_, const Complex* b_5, const Complex* c_5, bool dagger_,
          int parity, int shift_[4], int halo_shift_[4], const double scale_, MdwfFusedDslashType type_)
          : out(out)
          , in(in)
          , U(U)
          , y(y)
          , x(x)
          , nParity(in.SiteSubset())
          , parity(parity)
          , volume_cb(in.VolumeCB() > out.VolumeCB() ? in.VolumeCB() : out.VolumeCB())
          , volume_4d_cb(volume_cb / Ls_)
          , Ls(Ls_)
          , m_f(m_f_)
          , m_5(m_5_)
          , dagger(dagger_)
          , shift{shift_[0], shift_[1], shift_[2], shift_[3]}
          , halo_shift{halo_shift_[0], halo_shift_[1], halo_shift_[2], halo_shift_[3]}
          , dim{(3 - nParity) * (in.VolumeCB() > out.VolumeCB() ? in.X(0) : out.X(0)),
                in.VolumeCB() > out.VolumeCB() ? in.X(1) : out.X(1),
                in.VolumeCB() > out.VolumeCB() ? in.X(2) : out.X(2),
                in.VolumeCB() > out.VolumeCB() ? in.X(3) : out.X(3)}
          , shrinked_dim{dim[0] - 2 * shift[0], dim[1] - 2 * shift[1], dim[2] - 2 * shift[2], dim[3] - 2 * shift[3]}
          , volume_4d_cb_shift(shrinked_dim[0] * shrinked_dim[1] * shrinked_dim[2] * shrinked_dim[3] / 2)
          , scale(scale_)
          , type(type_) {
        if (in.Nspin() != 4) { errorQuda("nSpin = %d NOT supported.\n", in.Nspin()); }

        if (nParity == 2) { errorQuda("nParity = 2 NOT supported, yet.\n"); }

        if (!in.isNative() || !out.isNative())
          errorQuda("Unsupported field order out=%d in=%d\n", out.FieldOrder(), in.FieldOrder());

        b = b_5[0].real();
        c = c_5[0].real();
        kappa = -(c * (4. + m_5) - 1.) / (b * (4. + m_5) + 1.); // This is actually -kappa in my(Jiqun Tu) notes.
        fac_inv = 0.5
            / (1. + std::pow(kappa, (int)Ls) * m_f); // 0.5 to normalize the (1 +/- gamma5) in the chiral projector.
        switch (type) {
        case dslash4_dslash5pre_dslash5inv:
        case dslash4dag_dslash5predag_dslash5invdag:
          m_scale = b;
          alpha = 1. + c / (kappa * b); // b-c/kappa = b(1-c/(b*kappa))
          beta = -c / (kappa * b);
          break;
        case dslash4_dslash5inv_dslash5invdag:
          m_scale = -0.25 / ((b * (4. + m_5) + 1.) * (b * (4. + m_5) + 1.)); // -kappa_b^2
          break;
        case dslash4dag_dslash5predag:
          m_scale = -0.25 / ((b * (4. + m_5) + 1.) * (b * (4. + m_5) + 1.)) * b; // -kappa_b^2
          alpha = c / (2. * b); // 2 to compensate for the spin projection
          beta = 1.;
          break;
        case 4:
          m_scale = b;
          alpha = c / (2. * b);
          beta = 1.;
          break;
        default: errorQuda("Unknown MdwfFusedDslashType %d", type);
        }
      }
    };

    __device__ inline int index_4d_cb_from_coordinate_4d(const int coordinate[4], const int dim[4]) {
      return (((coordinate[3] * dim[2] + coordinate[2]) * dim[1] + coordinate[1]) * dim[0] + coordinate[0]) >> 1;
    }

    __device__ inline bool is_halo_4d(const int coordinate[4], const int dim[4], const int halo_shift[4]) {
      bool ret = false;
#pragma unroll
      for (int d = 0; d < 4; d++) {
        ret = ret or (coordinate[d] >= dim[d] - halo_shift[d] or coordinate[d] < halo_shift[d]);
      }
      return ret;
    }

    /**
    -> Everything should be understood in a 4d checkboarding sense.
    */
    template <class storage_type, bool dagger, bool halo, class Vector, class Arg>
    __device__ inline void apply_wilson_5d(Vector& out, int coordinate[4], Arg& arg, int s) {

      typedef typename mapper<storage_type>::type compute_type;
      typedef Matrix<complex<compute_type>, 3> Link;
      const int their_spinor_parity = arg.nParity == 2 ? 1 - arg.parity : 0;

      const int index_4d_cb = index_4d_cb_from_coordinate_4d(coordinate, arg.dim);

#pragma unroll
      for (int d = 0; d < 4; d++) // loop over dimension
      {
        coordinate[d]++;
        if (!halo || !is_halo_4d(coordinate, arg.dim, arg.halo_shift)) {
          // Forward gather - compute fwd offset for vector fetch
          const int fwd_idx = s * arg.volume_4d_cb + index_4d_cb_from_coordinate_4d(coordinate, arg.dim);
          constexpr int proj_dir = dagger ? +1 : -1;

          const Link U = arg.U(d, index_4d_cb, arg.parity);
          const Vector in = arg.in(fwd_idx, their_spinor_parity);
          out += (U * in.project(d, proj_dir)).reconstruct(d, proj_dir);
        }
        coordinate[d]-=2;
        if (!halo || !is_halo_4d(coordinate, arg.dim, arg.halo_shift)) {
          // Backward gather - compute back offset for spinor and gauge fetch
          const int gauge_idx = index_4d_cb_from_coordinate_4d(coordinate, arg.dim);
          ;
          const int back_idx = s * arg.volume_4d_cb + gauge_idx;
          constexpr int proj_dir = dagger ? -1 : +1;

          const Link U = arg.U(d, gauge_idx, 1 - arg.parity);
          const Vector in = arg.in(back_idx, their_spinor_parity);
          out += (conj(U) * in.project(d, proj_dir)).reconstruct(d, proj_dir);
        }
        coordinate[d]++;
      } // nDim
    }

    /**
    -> Everything should be understood in a 4d checkboarding sense.
    */
    template <class T>
    __device__ inline void coordinate_from_shrinked_index(int coordinate[4], int shrinked_index,
        const T shrinked_dim[4], const int shift[4], int parity) // s is the 5d stuff,
    {
      int aux[4];
      aux[0] = shrinked_index * 2;

#pragma unroll
      for (int i = 0; i < 3; i++) { aux[i + 1] = aux[i] / shrinked_dim[i]; }

      coordinate[0] = aux[0] - aux[1] * shrinked_dim[0];
      coordinate[1] = aux[1] - aux[2] * shrinked_dim[1];
      coordinate[2] = aux[2] - aux[3] * shrinked_dim[2];
      coordinate[3] = aux[3];

      // Find the full coordinate in the shrinked volume.
      coordinate[0] += (parity + coordinate[3] + coordinate[2] + coordinate[1]) & 1;

// Now go back to the extended volume.
#pragma unroll
      for (int d = 0; d < 4; d++) { coordinate[d] += shift[d]; }
    }
#if 0
    /**
      @brief Tensor core kernel for applying Wilson hopping term and then the beta + alpha*M5inv operator
      The kernels type(type_) will be specified in some documentations.
    */
    template <class storage_type, int block_dim_x, int Ls, int minBlocksPerMultiprocessor, bool reload, class Arg,
        int type_>
    __global__ void __launch_bounds__(block_dim_x*Ls, minBlocksPerMultiprocessor) fused_tensor_core(Arg arg)
    {
      const int explicit_parity = arg.nParity == 2 ? arg.parity : 0;

      TensorCoreSharedMemory<half2> shared_memory_data;

      constexpr int M = 4 * Ls;
      constexpr int N = 6 * block_dim_x;

      constexpr int sm_m_pad_size = 0;
      constexpr int sm_n_pad_size = 16;

      constexpr int N_sm = N + sm_n_pad_size;
      constexpr int M_sm = M + sm_m_pad_size;

      half2* sm_b = shared_memory_data;
      half* sm_c = reinterpret_cast<half*>(sm_b);

      half* sm_a = reload ? sm_c + M * N_sm : sm_c;
      // This is for type == 1 ONLY.
      half* sm_a_black = sm_a + M * M_sm;

      if (type_ == 0) {
        construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, false, Arg>(arg, sm_a); // dagger = false
      } else if (type_ == 2) {
        construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, true, Arg>(arg, sm_a); // dagger =  true
      } else if (type_ == 1) {
        construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, false, Arg>(arg, sm_a); // dagger = false
      } else if (type_ == 3) {
        construct_matrix_a_d5<block_dim_x, Ls, M_sm, true, Arg>(arg, sm_a); // dagger =  true
      } else if (type_ == 4) {
        construct_matrix_a_d5<block_dim_x, Ls, M_sm, false, Arg>(arg, sm_a); // dagger =  true
      }
      __syncthreads();

      bool idle = false;
      int s4_shift_base = blockIdx.x * blockDim.x; // base.
      int s4_shift, sid;

      constexpr int WMMA_M = 16;
      constexpr int WMMA_N = 16;
      constexpr int WMMA_K = 16;

      constexpr int tm_dim = M / WMMA_M;
      constexpr int tn_dim = N / WMMA_N;

      constexpr int total_warp = block_dim_x * Ls >> 5;
      const int this_warp = (threadIdx.y * block_dim_x + threadIdx.x) >> 5;

      constexpr int total_tile = tm_dim * tn_dim;

      constexpr int warp_cycle = total_tile / total_warp;
      const int warp_m = this_warp * warp_cycle / tn_dim;

      typedef
          typename nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::col_major>
              a_type;

      a_type a_frag[reload ? 1 : tm_dim];
      a_type a_frag_black[reload ? 1 : tm_dim];
      if (!reload) { // in the preload case we preload ...
#pragma unroll
        for (int k = 0; k < tm_dim; k++) {
          const int a_row = warp_m * WMMA_M;
          const int a_col = k * WMMA_K;
          // Load Matrix
          nvcuda::wmma::load_matrix_sync(a_frag[k], sm_a + a_row + a_col * M_sm, M_sm);
        }
      }

      if (type_ == 1) {
        arg.alpha = 1.;
        if (!reload) { // in the preload case we preload ...
          construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, true, Arg>(arg, sm_a); // dagger = true
          __syncthreads();
#pragma unroll
          for (int k = 0; k < tm_dim; k++) {
            const int a_row = warp_m * WMMA_M;
            const int a_col = k * WMMA_K;
            // Load Matrix
            nvcuda::wmma::load_matrix_sync(a_frag_black[k], sm_c + a_row + a_col * M_sm, M_sm);
          }
        } else {
          construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, true, Arg>(arg, sm_a_black); // dagger = true
          __syncthreads();
        }
      }

      while (s4_shift_base < arg.volume_4d_cb_shift) {
        int x[4];
        s4_shift = s4_shift_base + threadIdx.x;
        coordinate_from_shrinked_index(x, s4_shift, arg.shrinked_dim, arg.shift, arg.parity);
        sid = threadIdx.y * arg.volume_4d_cb + index_4d_cb_from_coordinate_4d(x, arg.dim);

        if (s4_shift >= arg.volume_4d_cb_shift) { idle = true; }

        typedef typename mapper<storage_type>::type real;
        typedef ColorSpinor<real, 3, 4> Vector;

        if (!idle) {
          Vector in_vec;
          // the Wilson hopping terms
          if (type_ == 0) {
            apply_wilson_5d<storage_type, false, true>(in_vec, x, arg, threadIdx.y); // dagger = false; halo =  true
          } else if (type_ == 2) {
            apply_wilson_5d<storage_type, true, false>(in_vec, x, arg, threadIdx.y); // dagger =  true; halo = false
          } else if (type_ == 1) {
            apply_wilson_5d<storage_type, false, true>(in_vec, x, arg, threadIdx.y); // dagger = false; halo =  true
          } else if (type_ == 3) {
            apply_wilson_5d<storage_type, true, false>(in_vec, x, arg, threadIdx.y); // dagger =  true; halo = false
          } else if (type_ == 4) {
            int sid_shift = threadIdx.y * arg.volume_4d_cb_shift + s4_shift;
            in_vec = arg.in(sid_shift, explicit_parity);
          }
          // store result to shared memory
          load_matrix_b_vector<N_sm / 2, false>(in_vec, sm_b, arg.scale); // acc(accumulation) = false
        }

        __syncthreads();
        // wmma_gemm<block_dim_x, Ls, M, N, M_sm, N_sm, reload>(a_frag, sm_a, sm_c, sm_c);
        mma_sync_gemm<block_dim_x, Ls, M, N, M_sm, N_sm>(sm_a, sm_c, sm_c);
        __syncthreads();

        if (type_ == 1) {

          if (!idle) {
            constexpr int in_x_shift = 2;
            int back_x[4] = {x[0] - in_x_shift, x[1] - in_x_shift, x[2] - in_x_shift, x[3] - in_x_shift};
            int back_dim[4] = {arg.dim[0] - in_x_shift*2, arg.dim[1] - in_x_shift*2, arg.dim[2] - in_x_shift*2, arg.dim[3] - in_x_shift*2};
            if (back_x[0] >= 0 && back_x[0] < back_dim[0] && back_x[1] >= 0 && back_x[1] < back_dim[1] && back_x[2] >= 0
                && back_x[2] < back_dim[2] && back_x[3] >= 0 && back_x[3] < back_dim[3]) {
              int volume_4d_cb_back = back_dim[0] * back_dim[1] * back_dim[2] * back_dim[3] >> 1;
              int sid_back_shift = threadIdx.y * volume_4d_cb_back + index_4d_cb_from_coordinate_4d(back_x, back_dim);
              Vector aux_in_vec = arg.x(sid_back_shift, explicit_parity);
              load_matrix_b_vector<N_sm / 2, true>(aux_in_vec, sm_b, arg.scale * arg.m_scale); // acc = true
            }
            store_matrix_c<storage_type, N_sm>(arg.y, sm_b, sid, arg.scale * arg.m_scale);
          }
          __syncthreads();
          wmma_gemm<block_dim_x, Ls, M, N, M_sm, N_sm, reload>(a_frag_black, sm_a_black, sm_c, sm_c);
          __syncthreads();

        } else if (type_ == 3) {

          if (!idle) {
            Vector aux_in_vec = arg.x(sid, explicit_parity);
            load_matrix_b_vector<N_sm / 2, true>(aux_in_vec, sm_b, arg.scale * arg.m_scale);
          }
        }

        if (type_ == 3) {
          if (!idle) {
            int sid_shift = threadIdx.y * arg.volume_4d_cb_shift + s4_shift;
            store_matrix_c<storage_type, N_sm>(arg.out, sm_b, sid_shift, arg.scale * arg.m_scale);
          }
        } else {
          if (!idle) { store_matrix_c<storage_type, N_sm>(arg.out, sm_b, sid, arg.scale * arg.m_scale); }
        }

        s4_shift_base += gridDim.x * blockDim.x;

      } // while
    }
#else
    /**
      @brief Tensor core kernel for applying Wilson hopping term and then the beta + alpha*M5inv operator
      The kernels type(type_) will be specified in some documentations.
    */
    template <class storage_type, int block_dim_x, int Ls, int minBlocksPerMultiprocessor, bool reload, class Arg,
        int type_>
    __global__ void __launch_bounds__(block_dim_x*Ls, minBlocksPerMultiprocessor) fused_tensor_core(Arg arg)
    {
      const int explicit_parity = arg.nParity == 2 ? arg.parity : 0;

      TensorCoreSharedMemory<half2> shared_memory_data;

      constexpr int M = 4 * Ls;
      constexpr int N = 6 * block_dim_x;

      constexpr int sm_m_pad_size = 0;
      constexpr int sm_n_pad_size = 16;

      constexpr int N_sm = N + sm_n_pad_size;
      constexpr int M_sm = M + sm_m_pad_size;

      half2* sm_b = shared_memory_data;
      half* sm_c = reinterpret_cast<half*>(sm_b);

      half* sm_a = sm_c + M * N_sm;
      // This is for type == 1 ONLY.
      half* sm_a_black = sm_a + M * M_sm;

      if (type_ == 0) {
        construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, false, Arg>(arg, sm_a); // dagger = false
      } else if (type_ == 2) {
        construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, true, Arg>(arg, sm_a); // dagger =  true
      } else if (type_ == 1) {
        construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, false, Arg>(arg, sm_a); // dagger = false
      } else if (type_ == 3) {
        construct_matrix_a_d5<block_dim_x, Ls, M_sm, true, Arg>(arg, sm_a); // dagger =  true
      } else if (type_ == 4) {
        construct_matrix_a_d5<block_dim_x, Ls, M_sm, false, Arg>(arg, sm_a); // dagger =  true
      }
      __syncthreads();

      bool idle = false;
      int s4_shift_base = blockIdx.x * blockDim.x; // base.
      int s4_shift, sid;

      if (type_ == 1) {
        arg.alpha = 1.;
        construct_matrix_a_m5inv<block_dim_x, Ls, M_sm, true, Arg>(arg, sm_a_black); // dagger = true
        __syncthreads();
      }

      while (s4_shift_base < arg.volume_4d_cb_shift) {
        int x[4];
        s4_shift = s4_shift_base + threadIdx.x;
        coordinate_from_shrinked_index(x, s4_shift, arg.shrinked_dim, arg.shift, arg.parity);
        sid = threadIdx.y * arg.volume_4d_cb + index_4d_cb_from_coordinate_4d(x, arg.dim);

        if (s4_shift >= arg.volume_4d_cb_shift) { idle = true; }

        typedef typename mapper<storage_type>::type real;
        typedef ColorSpinor<real, 3, 4> Vector;

        if (!idle) {
          Vector in_vec;
          // the Wilson hopping terms
          if (type_ == 0) {
            apply_wilson_5d<storage_type, false, true>(in_vec, x, arg, threadIdx.y); // dagger = false; halo =  true
          } else if (type_ == 2) {
            apply_wilson_5d<storage_type, true, false>(in_vec, x, arg, threadIdx.y); // dagger =  true; halo = false
          } else if (type_ == 1) {
            apply_wilson_5d<storage_type, false, true>(in_vec, x, arg, threadIdx.y); // dagger = false; halo =  true
          } else if (type_ == 3) {
            apply_wilson_5d<storage_type, true, false>(in_vec, x, arg, threadIdx.y); // dagger =  true; halo = false
          } else if (type_ == 4) {
            int sid_shift = threadIdx.y * arg.volume_4d_cb_shift + s4_shift;
            in_vec = arg.in(sid_shift, explicit_parity);
          }
          // store result to shared memory
          load_matrix_b_vector<N_sm / 2, false>(in_vec, sm_b, arg.scale); // acc(accumulation) = false
        }

        __syncthreads();
        mma_sync_gemm<block_dim_x, Ls, M, N, M_sm, N_sm>(sm_a, sm_c, sm_c);
        __syncthreads();

        if (type_ == 1) {

          if (!idle) {
            constexpr int in_x_shift = 2;
            int back_x[4] = {x[0] - in_x_shift, x[1] - in_x_shift, x[2] - in_x_shift, x[3] - in_x_shift};
            int back_dim[4] = {arg.dim[0] - in_x_shift*2, arg.dim[1] - in_x_shift*2, arg.dim[2] - in_x_shift*2, arg.dim[3] - in_x_shift*2};
            if (back_x[0] >= 0 && back_x[0] < back_dim[0] && back_x[1] >= 0 && back_x[1] < back_dim[1] && back_x[2] >= 0
                && back_x[2] < back_dim[2] && back_x[3] >= 0 && back_x[3] < back_dim[3]) {
              int volume_4d_cb_back = back_dim[0] * back_dim[1] * back_dim[2] * back_dim[3] >> 1;
              int sid_back_shift = threadIdx.y * volume_4d_cb_back + index_4d_cb_from_coordinate_4d(back_x, back_dim);
              Vector aux_in_vec = arg.x(sid_back_shift, explicit_parity);
              load_matrix_b_vector<N_sm / 2, true>(aux_in_vec, sm_b, arg.scale * arg.m_scale); // acc = true
            }
            store_matrix_c<storage_type, N_sm>(arg.y, sm_b, sid, arg.scale * arg.m_scale);
          }
          __syncthreads();
          mma_sync_gemm<block_dim_x, Ls, M, N, M_sm, N_sm>(sm_a_black, sm_c, sm_c);
          __syncthreads();

        } else if (type_ == 3) {

          if (!idle) {
            Vector aux_in_vec = arg.x(sid, explicit_parity);
            load_matrix_b_vector<N_sm / 2, true>(aux_in_vec, sm_b, arg.scale * arg.m_scale);
          }
        }

        if (type_ == 3) {
          if (!idle) {
            int sid_shift = threadIdx.y * arg.volume_4d_cb_shift + s4_shift;
            store_matrix_c<storage_type, N_sm>(arg.out, sm_b, sid_shift, arg.scale * arg.m_scale);
          }
        } else {
          if (!idle) { store_matrix_c<storage_type, N_sm>(arg.out, sm_b, sid, arg.scale * arg.m_scale); }
        }

        s4_shift_base += gridDim.x * blockDim.x;

      } // while
    }
#endif
    template <class storage_type, int Ls_, class Arg> class FusedDslash : public TunableVectorYZ {

  protected:
      Arg& arg;
      const ColorSpinorField& meta;
      static constexpr bool shared = true; // whether to use shared memory cache blocking for M5inv

      /** Whether to use variable or fixed coefficient algorithm.  Must be true if using ZMOBIUS */
      static constexpr bool var_inverse = true;

      long long flops() const {
        constexpr long long hop = 7ll * 8ll; // 8 for eight directions, 7 comes from Peter/Grid's count
        constexpr long long mat = 2ll * 4ll * Ls_ - 1ll;
        long long volume_4d_cb_halo_shift = (arg.dim[0] - 2 * arg.halo_shift[0]) * (arg.dim[1] - 2 * arg.halo_shift[1])
            * (arg.dim[2] - 2 * arg.halo_shift[2]) * (arg.dim[3] - 2 * arg.halo_shift[3]) / 2;

        long long flops_ = 0;
        switch (arg.type) {
        // I am too lazy to fix the flops count. :(
        case 0: // FIXME: flops
          flops_ = volume_4d_cb_halo_shift * 6ll * 4ll * Ls_ * hop + arg.volume_4d_cb_shift * 24ll * Ls_ * mat;
          break;
        case 1:
          flops_ = volume_4d_cb_halo_shift * 6ll * 4ll * Ls_ * hop + arg.volume_4d_cb_shift * 24ll * Ls_ * 2ll * mat;
          break;
        case 2:
        case 3:
          flops_ = arg.volume_4d_cb_shift * 6ll * 4ll * Ls_
              * (hop + mat); // for 2 and 3 we don't have the halo complication.
          break;
        case 4: flops_ = arg.volume_4d_cb_shift * 6ll * 4ll * Ls_ * (mat); break;
        default: errorQuda("Unknown MdwfFusedDslashType %d", arg.type);
        }

        return flops_;
      }

      long long bytes() const {
        switch (arg.type) {
        case 0: return arg.out.Bytes() + arg.in.Bytes() + arg.U.Bytes();
        case 1: return 3 * arg.out.Bytes() + arg.in.Bytes() + arg.U.Bytes();
        case 2: return arg.out.Bytes() + arg.in.Bytes() + arg.U.Bytes();
        case 3:
        case 4: return 2 * arg.out.Bytes() + arg.in.Bytes() + arg.U.Bytes();
        default: errorQuda("Unknown MdwfFusedDslashType %d", arg.type);
        }
        return 0ll;
      }

      virtual bool tuneGridDim() const { return true; }
      virtual bool tuneAuxDim() const { return true; }
      virtual bool tuneSharedBytes() const { return true; }
      unsigned int minThreads() const { return arg.volume_4d_cb; }

      unsigned int shared_bytes_per_block(const TuneParam& param) const {
        // (Ls*4) by (Ls*4), (Ls*4) by (volume_4d*6 + 16)
        // if (param.aux.x == 1) { // aux.x == 1 --> reload == true
        if (true) { // aux.x == 1 --> reload == true
          if (arg.type == 1) {
            return ((param.block.y * 4) * (param.block.y * 4 + 0) * 2 + (param.block.y * 4) * (param.block.x * 6 + 16))
                * sizeof(half);
          } else {
            return ((param.block.y * 4) * (param.block.y * 4 + 0) + (param.block.y * 4) * (param.block.x * 6 + 16))
                * sizeof(half);
          }
        } else {
          int a_size = (param.block.y * 4) * (param.block.y * 4 + 0);
          int b_size = (param.block.y * 4) * (param.block.x * 6 + 16);
          return (a_size > b_size ? a_size : b_size) * sizeof(half);
        }
      }

      virtual bool advanceBlockDim(TuneParam& param) const {
        if (param.block.x < max_block_size()) {
          param.block.x += step_block_size();
          param.shared_bytes = shared_bytes_per_block(param);
          return true;
        } else {
          return false;
        }
      }

      virtual bool advanceGridDim(TuneParam& param) const {
        const unsigned int max_blocks = maxGridSize();
        const int step = deviceProp.multiProcessorCount;
        param.grid.x += step;
        if (param.grid.x > max_blocks) {
          return false;
        } else {
          param.block.x = min_block_size();
          param.shared_bytes = shared_bytes_per_block(param);
          return true;
        }
      }

      virtual bool advanceAux(TuneParam& param) const {
        bool aux_advanced = false;
        if (param.aux.x == 0) { // first see if aux.x(ONLY 0(false) or 1(true))
          param.aux.x++;
          aux_advanced = true;
        } else {
          if (param.aux.y < 3) { // second see if aux.y
            param.aux.y++;
            aux_advanced = true;
            param.aux.x = 0;
          }
        }
        if (aux_advanced) {
          // We have updated the "aux" so reset all other parameters.
          param.grid.x = minGridSize();
          param.block.x = min_block_size();
          param.shared_bytes = shared_bytes_per_block(param);
          return true;
        } else {
          return false;
        }
      }

      virtual unsigned int maxGridSize() const { return 32 * deviceProp.multiProcessorCount; }
      virtual unsigned int minGridSize() const { return deviceProp.multiProcessorCount; }
      unsigned int min_block_size() const { return 8; }
      unsigned int max_block_size() const { return 32; }
      unsigned int step_block_size() const { return 8; }

      // overloaded to return max dynamic shared memory if doing shared-memory inverse
      unsigned int maxSharedBytesPerBlock() const {
        if (shared) {
          return maxDynamicSharedBytesPerBlock();
        } else {
          return TunableVectorYZ::maxSharedBytesPerBlock();
        }
      }

  public:
      FusedDslash(Arg& arg, const ColorSpinorField& meta) : TunableVectorYZ(arg.Ls, arg.nParity), arg(arg), meta(meta) {
        strcpy(aux, meta.AuxString());
        if (arg.dagger) strcat(aux, ",Dagger");
        //        if (arg.xpay) strcat(aux,",xpay");
        char config[512];
        switch (arg.type) {
        case dslash4_dslash5pre_dslash5inv:
          sprintf(config, ",f0,shift%d,%d,%d,%d,halo%d,%d,%d,%d", arg.shift[0], arg.shift[1], arg.shift[2],
              arg.shift[3], arg.halo_shift[0], arg.halo_shift[1], arg.halo_shift[2], arg.halo_shift[3]);
          strcat(aux, config);
          break;
        case dslash4dag_dslash5predag_dslash5invdag:
          sprintf(config, ",f2,shift%d,%d,%d,%d", arg.shift[0], arg.shift[1], arg.shift[2], arg.shift[3]);
          strcat(aux, config);
          break;
        case dslash4_dslash5inv_dslash5invdag:
          sprintf(config, ",f1,shift%d,%d,%d,%d,halo%d,%d,%d,%d", arg.shift[0], arg.shift[1], arg.shift[2],
              arg.shift[3], arg.halo_shift[0], arg.halo_shift[1], arg.halo_shift[2], arg.halo_shift[3]);
          strcat(aux, config);
          break;
        case dslash4dag_dslash5predag:
          sprintf(config, ",f3,shift%d,%d,%d,%d", arg.shift[0], arg.shift[1], arg.shift[2], arg.shift[3]);
          strcat(aux, config);
          break;
        case 4:
          sprintf(config, ",f4,shift%d,%d,%d,%d", arg.shift[0], arg.shift[1], arg.shift[2], arg.shift[3]);
          strcat(aux, config);
          break;
        default: errorQuda("Unknown MdwfFusedDslashType %d", arg.type);
        }
      }
      virtual ~FusedDslash() {}

      template <typename T> inline void launch(T* f, const TuneParam& tp, Arg& arg, const hipStream_t& stream) {
        if (shared) { setMaxDynamicSharedBytesPerBlock(f); }
        void* args[] = {&arg};
        qudaLaunchKernel((const void*)f, tp.grid, tp.block, args, tp.shared_bytes, stream);
      }

      // The following apply<...> functions are used to turn the tune parameters into template arguments.
      // Specifically tp.aux.y dictates the minBlocksPerMultiprocessor in __launch_bounds__(..).
      // tp.aux.x dictates whether or not to reload.
      template <int block_dim_x, bool reload, int type>
      void apply(const TuneParam& tp, Arg& arg, const hipStream_t& stream) {
        switch (tp.aux.y) {
        case 1: launch(fused_tensor_core<storage_type, block_dim_x, Ls_, 1, reload, Arg, type>, tp, arg, stream); break;
        case 2: launch(fused_tensor_core<storage_type, block_dim_x, Ls_, 2, reload, Arg, type>, tp, arg, stream); break;
        case 3: launch(fused_tensor_core<storage_type, block_dim_x, Ls_, 3, reload, Arg, type>, tp, arg, stream); break;
        default: errorQuda("NOT valid tp.aux.y(=%d)\n", tp.aux.y);
        }
      }

      template <bool reload, int type> void apply(const TuneParam& tp, Arg& arg, const hipStream_t& stream) {
        switch (tp.block.x) {
          case  8: apply< 8, reload, type>(tp, arg, stream); break;
          case 16: apply<16, reload, type>(tp, arg, stream); break;
          case 24: apply<24, reload, type>(tp, arg, stream); break;
          case 32: apply<32, reload, type>(tp, arg, stream); break;
          default: errorQuda("NOT valid tp.block.x(=%d)\n", tp.block.x);
        }
      }

      template <int type> void apply(const TuneParam& tp, Arg& arg, const hipStream_t& stream) {
        if (tp.aux.x == 0) {
          apply<false, type>(tp, arg, stream); // reload = false
        } else {
          apply<true, type>(tp, arg, stream); // reload = true
        }
      }

      void apply(const hipStream_t& stream) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        switch (arg.type) {
          case 0: apply<0>(tp, arg, stream); break;
          case 1: apply<1>(tp, arg, stream); break;
          case 2: apply<2>(tp, arg, stream); break;
          case 3: apply<3>(tp, arg, stream); break;
          case 4: apply<4>(tp, arg, stream); break;
        default: errorQuda("Unknown MdwfFusedDslashType %d", arg.type);
        }
      }

      void initTuneParam(TuneParam& param) const {
        TunableVectorYZ::initTuneParam(param);
        param.block = dim3(min_block_size(), arg.Ls, 1); // Ls must be contained in the block
        param.grid = dim3(minGridSize(), 1, 1);
        param.shared_bytes = shared_bytes_per_block(param);
        param.aux.x = 0;
        param.aux.y = 1;
      }

      void defaultTuneParam(TuneParam& param) const { initTuneParam(param); }

      TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
    };

    // Apply the 5th dimension dslash operator to a colorspinor field
    // out = Dslash5 * in
    template <class storage_type>
    void apply_fused_dslash(ColorSpinorField& out, const ColorSpinorField& in, const GaugeField& U, ColorSpinorField& y,
        const ColorSpinorField& x, double m_f, double m_5, const Complex* b_5, const Complex* c_5, bool dagger,
        int parity, int shift[4], int halo_shift[4], const double scale, MdwfFusedDslashType type) {
      // switch for Ls
      switch (in.X(4)) {
      //      case  8:
      //        {
      //          FusedDslashArg<storage_type,  8> arg(out, in, U, y, x, m_f, m_5, b_5, c_5, dagger, parity, shift,
      //          halo_shift, scale, type);
      //          FusedDslash<storage_type,  8, FusedDslashArg<storage_type,  8> > dslash(arg, in);
      //          dslash.apply(streams[Nstream-1]);
      //        }
      //      break;
        case 12: {
          FusedDslashArg<storage_type, 12> arg(
              out, in, U, y, x, m_f, m_5, b_5, c_5, dagger, parity, shift, halo_shift, scale, type);
          FusedDslash<storage_type, 12, FusedDslashArg<storage_type, 12>> dslash(arg, in);
          dslash.apply(streams[Nstream - 1]);
        } break;
      //      case 16:
      //        {
      //          FusedDslashArg<storage_type, 16> arg(out, in, U, y, x, m_f, m_5, b_5, c_5, dagger, parity, shift,
      //          halo_shift, scale, type);
      //          FusedDslash<storage_type, 16, FusedDslashArg<storage_type, 16> > dslash(arg, in);
      //          dslash.apply(streams[Nstream-1]);
      //        }
      //      break;
      //      case 20:
      //        {
      //          FusedDslashArg<storage_type, 20> arg(out, in, U, y, x, m_f, m_5, b_5, c_5, dagger, parity, shift,
      //          halo_shift, scale, type);
      //          FusedDslash<storage_type, 20, FusedDslashArg<storage_type, 20> > dslash(arg, in);
      //          dslash.apply(streams[Nstream-1]);
      //        }
      //      break;
      //      case 24:
      //        {
      //          FusedDslashArg<storage_type, 24> arg(out, in, U, y, x, m_f, m_5, b_5, c_5, dagger, parity, shift,
      //          halo_shift, scale, type);
      //          FusedDslash<storage_type, 24, FusedDslashArg<storage_type, 24> > dslash(arg, in);
      //          dslash.apply(streams[Nstream-1]);
      //        }
      //      break;
      default: errorQuda("Ls = %d is NOT supported.\n", in.X(4));
      }
    }
#endif // defined (GPU_DOMAIN_WALL_DIRAC) && (__COMPUTE_CAPABILITY__ >= 700)

    void apply_fused_dslash(ColorSpinorField& out, const ColorSpinorField& in, const GaugeField& U, ColorSpinorField& y,
        const ColorSpinorField& x, double m_f, double m_5, const Complex* b_5, const Complex* c_5, bool dagger,
        int parity, int shift[4], int halo_shift[4], const double scale, MdwfFusedDslashType type) {
#if defined(GPU_DOMAIN_WALL_DIRAC) && (__COMPUTE_CAPABILITY__ >= 700)
      checkLocation(out, in); // check all locations match

      if (checkPrecision(out, in) == QUDA_HALF_PRECISION && in.Ncolor() == 3) {
        apply_fused_dslash<short>(out, in, U, y, x, m_f, m_5, b_5, c_5, dagger, parity, shift, halo_shift, scale, type);
      } else if (checkPrecision(out, in) == QUDA_QUARTER_PRECISION && in.Ncolor() == 3) {
        apply_fused_dslash<char>(out, in, U, y, x, m_f, m_5, b_5, c_5, dagger, parity, shift, halo_shift, scale, type);
      } else {
        errorQuda("Tensor core implemtation ONLY supports HALF/QUARTER precision and n_color = 3.\n");
      }

#else
      errorQuda("Domain wall dslash WITH tensor cores has not been built");
#endif
    }
  } // namespace mobius_tensor_core
} // namespace quda
